#include "hip/hip_runtime.h"
// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################

#include <iostream>
#include <string>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include "helper.cuh"
#include "gradient.cuh"
#include "divergence.cuh"
#include "norm.cuh"


int main(int argc,char **argv)
{
    // parse command line parameters
    const char *params = {
        "{i|image| |input image}"
        "{b|bw|false|load input image as grayscale/black-white}"
        "{r|repeats|1|number of computation repetitions}"
    };
    cv::CommandLineParser cmd(argc, argv, params);

    // input image
    std::string inputImage = cmd.get<std::string>("image");
    // number of computation repetitions to get a better run time measurement
    size_t repeats = (size_t)cmd.get<int>("repeats");
    // load the input image as grayscale
    bool gray = cmd.get<bool>("bw");

    // init camera
    bool useCam = inputImage.empty();
    cv::VideoCapture camera;
    if (useCam && !openCamera(camera, 0))
    {
        std::cerr << "ERROR: Could not open camera" << std::endl;
        return 1;
    }

    // read input frame
    cv::Mat mIn;
    if (useCam)
    {
        // read in first frame to get the dimensions
        camera >> mIn;
    }
    else
    {
        // load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
        mIn = cv::imread(inputImage.c_str(), (gray ? CV_LOAD_IMAGE_GRAYSCALE : -1));
    }
    // check
    if (mIn.empty())
    {
        std::cerr << "ERROR: Could not retrieve frame " << inputImage << std::endl;
        return 1;
    }
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn, CV_32F);

    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    std::cout << "Image: " << w << " x " << h << std::endl;

    // initialize CUDA context
    hipDeviceSynchronize();  CUDA_CHECK;

    // ### Set the output image format
    cv::Mat mOut_lapNorm(h,w,CV_32FC1);    // grayscale, 1 layer
    cv::Mat mOut_u(h,w,mIn.type());    // rgb, 1 layer
    cv::Mat mOut_v(h,w,mIn.type());    // rgb, 1 layer
    cv::Mat mOut_w(h,w,mIn.type());    // rgb, 1 layer

    // ### Allocate arrays
    // allocate raw input image array
    float *imgIn = NULL;    // TODO allocate array
    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut_lapNorm = NULL;   // TODO allocate array
    float *imgOut_u = NULL;         // TODO allocate array
    float *imgOut_v = NULL;         // TODO allocate array
    float *imgOut_w = NULL;         // TODO allocate array

    // allocate arrays on GPU
    float *d_imgIn = NULL;
    float *d_lapNorm = NULL;
    float *d_u = NULL;
    float *d_v = NULL;
    float *d_w = NULL;
    // TODO alloc cuda memory for device arrays

    do
    {
        // convert range of each channel to [0,1]
        mIn /= 255.0f;

        // init raw input image array (and convert to layered)
        convertMatToLayered (imgIn, mIn);
        // upload to GPU
        // TODO copy from imgIn to d_imgIn

        Timer timer;
        timer.start();
        for(size_t i = 0; i < repeats; ++i)
        {
            // TODO (4.1) implement computeGradientCuda() in gradient.cu
            computeGradientCuda(d_u, d_v, d_imgIn, w, h, nc);
            hipDeviceSynchronize();

            // TODO (4.2) implement computeDivergenceCuda() in divergence.cu
            computeDivergenceCuda(d_w, d_u, d_v, w, h, nc);
            hipDeviceSynchronize();

            // TODO (4.3) implement computeNormCuda() in norm.cu
            computeNormCuda(d_lapNorm, d_w, w, h, nc);
            hipDeviceSynchronize();
        }
        timer.end();
        float t = timer.get()/repeats;
        std::cout << "average time: " << t*1000 << " ms" << std::endl;

        // copy back to CPU
        // TODO download from device arrays to host arrays

        // show input image
        showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

        // show output image: first convert to interleaved opencv format from the layered raw array
        // TODO (4.4) show gradient, divergence and laplacian

        if (useCam)
        {
            // wait 30ms for key input
            if (cv::waitKey(30) >= 0)
            {
                mIn.release();
            }
            else
            {
                // retrieve next frame from camera
                camera >> mIn;
                // convert to float representation (opencv loads image values as single bytes by default)
                mIn.convertTo(mIn, CV_32F);
            }
        }
    }
    while (useCam && !mIn.empty());

    if (!useCam)
    {
        cv::waitKey(0);

        // save input and result
        //cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
        cv::imwrite("image_result.png",mOut_lapNorm*255.f);
    }

    // ### Free allocated arrays
    // TODO free cuda memory of all device arrays
    // TODO free memory of all host arrays

    // close all opencv windows
    cv::destroyAllWindows();

    return 0;
}
