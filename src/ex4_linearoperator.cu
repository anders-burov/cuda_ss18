#include "hip/hip_runtime.h"
// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################

#include <iostream>
#include <string>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include "helper.cuh"
#include "gradient.cuh"
#include "divergence.cuh"
#include "norm.cuh"


int main(int argc,char **argv)
{
    // parse command line parameters
    const char *params = {
        "{i|image| |input image}"
        "{b|bw|false|load input image as grayscale/black-white}"
        "{r|repeats|1|number of computation repetitions}"
    };
    cv::CommandLineParser cmd(argc, argv, params);

    // input image
    std::string inputImage = cmd.get<std::string>("image");
    // number of computation repetitions to get a better run time measurement
    size_t repeats = (size_t)cmd.get<int>("repeats");
    // load the input image as grayscale
    bool gray = cmd.get<bool>("bw");

    // init camera
    bool useCam = inputImage.empty();
    cv::VideoCapture camera;
    if (useCam && !openCamera(camera, 0))
    {
        std::cerr << "ERROR: Could not open camera" << std::endl;
        return 1;
    }

    // read input frame
    cv::Mat mIn;
    if (useCam)
    {
        // read in first frame to get the dimensions
        camera >> mIn;
    }
    else
    {
        // load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
        mIn = cv::imread(inputImage.c_str(), (gray ? CV_LOAD_IMAGE_GRAYSCALE : -1));
    }
    // check
    if (mIn.empty())
    {
        std::cerr << "ERROR: Could not retrieve frame " << inputImage << std::endl;
        return 1;
    }
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn, CV_32F);

    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    int n = w*h*nc;
    std::cout << "Image: " << w << " x " << h << std::endl;

    // initialize CUDA context
    hipDeviceSynchronize();  CUDA_CHECK;

    // ### Set the output image format
    cv::Mat mOut_lapNorm(h,w,CV_32FC1);    // grayscale, 1 layer
    cv::Mat mOut_u(h,w,mIn.type());    // rgb, 1 layer
    cv::Mat mOut_v(h,w,mIn.type());    // rgb, 1 layer
    cv::Mat mOut_w(h,w,mIn.type());    // rgb, 1 layer

    // ### Allocate arrays
    // allocate raw input image array
    float *imgIn = new float[n];
    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut_lapNorm = new float[w*h];
    float *imgOut_u = new float[n];
    float *imgOut_v = new float[n];
    float *imgOut_w = new float[n];

    // allocate arrays on GPU
    float *d_imgIn = NULL;
    float *d_lapNorm = NULL;
    float *d_u = NULL;
    float *d_v = NULL;
    float *d_w = NULL;
    // TODO alloc cuda memory for device arrays
    hipMalloc(&d_imgIn, n* sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_lapNorm, w*h* sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_u, n* sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_v, n* sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_w, n* sizeof(float)); CUDA_CHECK;

    do
    {
        // convert range of each channel to [0,1]
        mIn /= 255.0f;

        // init raw input image array (and convert to layered)
        convertMatToLayered (imgIn, mIn);
        // upload to GPU
        // TODO copy from imgIn to d_imgIn
        hipMemcpy(d_imgIn, imgIn, n * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;

        Timer timer;
        timer.start();
        for(size_t i = 0; i < repeats; ++i)
        {
            // TODO (4.1) implement computeGradientCuda() in gradient.cu
            computeGradientCuda(d_u, d_v, d_imgIn, w, h, nc);
            hipDeviceSynchronize();

            // TODO (4.2) implement computeDivergenceCuda() in divergence.cu
            computeDivergenceCuda(d_w, d_u, d_v, w, h, nc);
            hipDeviceSynchronize();

            // TODO (4.3) implement computeNormCuda() in norm.cu
            computeNormCuda(d_lapNorm, d_w, w, h, nc);
            hipDeviceSynchronize();
        }
        timer.end();
        float t = timer.get()/repeats;
        std::cout << "average time: " << t*1000 << " ms" << std::endl;

        // copy back to CPU
        // TODO download from device arrays to host arrays
        hipMemcpy(imgOut_lapNorm, d_lapNorm, w*h * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
        hipMemcpy(imgOut_u, d_u, n * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
        hipMemcpy(imgOut_v, d_v, n * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
        hipMemcpy(imgOut_w, d_w, n * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

        // show input image
        showImage("Input", mIn, 100, 80);  // show at position (x_from_left=100,y_from_above=100)

        // show output image: first convert to interleaved opencv format from the layered raw array
        // TODO (4.4) show gradient, divergence and laplacian
        convertLayeredToMat(mOut_u, imgOut_u);
        convertLayeredToMat(mOut_v, imgOut_v);
        convertLayeredToMat(mOut_w, imgOut_w);
        convertLayeredToMat(mOut_lapNorm, imgOut_lapNorm);
        showImage("gradient x", mOut_u*10, 100, 80+h+40);
        showImage("gradient y", mOut_v*10, 100+w/2 + 40, 80+h+40);
        showImage("divergence", mOut_w*10, 100+w/2 + 40, 80+h);
        showImage("laplacian per channel", mOut_w, 100 + w + 80, 80+h+40);
        showImage("laplacian norm", mOut_lapNorm, 100 + w + 80, 80);

        if (useCam)
        {
            // wait 30ms for key input
            if (cv::waitKey(30) >= 0)
            {
                mIn.release();
            }
            else
            {
                // retrieve next frame from camera
                camera >> mIn;
                // convert to float representation (opencv loads image values as single bytes by default)
                mIn.convertTo(mIn, CV_32F);
            }
        }
    }
    while (useCam && !mIn.empty());

    if (!useCam)
    {
        cv::waitKey(0);

        // save input and result
        //cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
        cv::imwrite("image_result.png",mOut_lapNorm*255.f);
    }

    // ### Free allocated arrays
    // TODO free cuda memory of all device arrays
    hipFree(d_imgIn); CUDA_CHECK;
    hipFree(d_lapNorm); CUDA_CHECK;
    hipFree(d_u); CUDA_CHECK;
    hipFree(d_v); CUDA_CHECK;
    hipFree(d_w); CUDA_CHECK;

    // TODO free memory of all host arrays
    delete[] imgIn;
    delete[] imgOut_lapNorm;
    delete[] imgOut_u;
    delete[] imgOut_v;
    delete[] imgOut_w;

    // close all opencv windows
    cv::destroyAllWindows();

    return 0;
}
