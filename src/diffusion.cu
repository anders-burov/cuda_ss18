#include "hip/hip_runtime.h"
// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################
#include "diffusion.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include "helper.cuh"

__global__
void updateDiffusivityKernel(float *u, const float *d_div, int w, int h, int nc, float dt)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    // TODO (9.5) update diffusivity
    if (x >= w || y >= h) return;

    u[y*w + x] += dt * d_div[y*w + x];
}


__global__
void multDiffusivityKernel(float *v1, float *v2, int w, int h, int nc, float epsilon)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    // TODO (9.3) multiply diffusivity
    if (x >= w || y >= h) return;

    float g = 0;
    for (int z = 0; z < nc; z++)
    {
        g += v1[z*h*w + y*w + x]*v1[z*h*w + y*w + x];
        g += v2[z*h*w + y*w + x]*v2[z*h*w + y*w + x];
    }
    g = 1/max(epsilon,sqrtf(g));

    for (int z = 0; z < nc; z++)
    {
        v1[z*h*w + y*w + x] *= g;
        v2[z*h*w + y*w + x] *= g;
    }
}

__global__
void multDiffusivityAnisotropicKernel(float *v1, float *v2, float *g11, float *g12, float *g22, int w, int h, int nc)
{
    // TODO (10.2) multiply diffusivity (anisotropic)
}


__global__
void computeDiffusivityKernel(float *diffusivity, const float *u, int w, int h, int nc, float epsilon)
{
    // TODO (11.2) compute diffusivity
}


__device__ void eigen(float a11, float a12, float a22, float* lmin, float* lmax, float* v11, float* v12, float* v21, float* v22)
{
    // TODO (10.1) compute eigen values and eigen vectors
}


__global__
void computeDiffusionTensorKernel(float *d_difftensor11, float *d_difftensor12, float *d_difftensor22,
                                  float *d_tensor11, float *d_tensor12, float *d_tensor22,
                                  float alpha, float C, int w, int h, int nc)
{
    // TODO (10.1) compute diffusion tensor
}


void updateDiffusivityCuda(float *u, const float *d_div, int w, int h, int nc, float dt)
{
    // calculate block and grid size
    dim3 block(32, 8, nc);     // TODO (9.5) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // run cuda kernel
    // TODO (9.5) execute kernel for updating diffusivity
    updateDiffusivityKernel <<<grid, block>>> (u, d_div, w, h, nc, dt);

    // check for errors
    // TODO (9.5)
    CUDA_CHECK;
}


void multDiffusivityCuda(float *v1, float *v2, int w, int h, int nc, float epsilon)
{
    // calculate block and grid size
    dim3 block(32, 8, 1);     // TODO (9.3) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // run cuda kernel
    // TODO (9.3) execute kernel for multiplying diffusivity
    multDiffusivityKernel <<<grid, block>>> (v1, v2, w, h, nc, epsilon);

    // check for errors
    // TODO (9.3)
    CUDA_CHECK;
}


void multDiffusivityAnisotropicCuda(float *v1, float *v2, float *g11, float *g12, float *g22, int w, int h, int nc)
{
    // calculate block and grid size
    dim3 block(0, 0, 0);     // TODO (10.2) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // run cuda kernel
    // TODO (10.2) execute kernel for multiplying diffusivity (anisotropic)

    // check for errors
    // TODO (10.2)
}


void computeDiffusivityCuda(float *diffusivity, const float *u, int w, int h, int nc, float epsilon)
{
    // calculate block and grid size
    dim3 block(0, 0, 0);     // TODO (11.2) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // run cuda kernel
    // TODO (11.2) execute kernel for computing diffusivity

    // check for errors
    // TODO (11.2)
}


void computeDiffusionTensorCuda(float *d_difftensor11, float *d_difftensor12, float *d_difftensor22,
                                float *d_tensor11, float *d_tensor12, float *d_tensor22,
                                float alpha, float C, int w, int h, int nc)
{
    // calculate block and grid size
    dim3 block(0, 0, 0);     // TODO (10.1) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // run cuda kernel
    // TODO (10.1) execute kernel for computing diffusion tensor

    // check for errors
    // TODO (10.1)
}
