#include "hip/hip_runtime.h"
// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################
#include "diffusion.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include "helper.cuh"


__global__
void updateDiffusivityKernel(float *u, const float *d_div, int w, int h, int nc, float dt)
{
    // TODO (9.5) update diffusivity
}


__global__
void multDiffusivityKernel(float *v1, float *v2, int w, int h, int nc, float epsilon)
{
    // TODO (9.3) multiply diffusivity
}

__global__
void multDiffusivityAnisotropicKernel(float *v1, float *v2, float *g11, float *g12, float *g22, int w, int h, int nc)
{
    // TODO (10.2) multiply diffusivity (anisotropic)
}


__global__
void computeDiffusivityKernel(float *diffusivity, const float *u, int w, int h, int nc, float epsilon)
{
    // TODO (11.2) compute diffusivity
}


__device__ void eigen(float a11, float a12, float a22, float* lmin, float* lmax, float* v11, float* v12, float* v21, float* v22)
{
    // TODO (10.1) compute eigen values and eigen vectors
}


__global__
void computeDiffusionTensorKernel(float *d_difftensor11, float *d_difftensor12, float *d_difftensor22,
                                  float *d_tensor11, float *d_tensor12, float *d_tensor22,
                                  float alpha, float C, int w, int h, int nc)
{
    // TODO (10.1) compute diffusion tensor
}


void updateDiffusivityCuda(float *u, const float *d_div, int w, int h, int nc, float dt)
{
    // calculate block and grid size
    dim3 block(0, 0, 0);     // TODO (9.5) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // run cuda kernel
    // TODO (9.5) execute kernel for updating diffusivity

    // check for errors
    // TODO (9.5)
}


void multDiffusivityCuda(float *v1, float *v2, int w, int h, int nc, float epsilon)
{
    // calculate block and grid size
    dim3 block(0, 0, 0);     // TODO (9.3) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // run cuda kernel
    // TODO (9.3) execute kernel for multiplying diffusivity

    // check for errors
    // TODO (9.3)
}


void multDiffusivityAnisotropicCuda(float *v1, float *v2, float *g11, float *g12, float *g22, int w, int h, int nc)
{
    // calculate block and grid size
    dim3 block(0, 0, 0);     // TODO (10.2) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // run cuda kernel
    // TODO (10.2) execute kernel for multiplying diffusivity (anisotropic)

    // check for errors
    // TODO (10.2)
}


void computeDiffusivityCuda(float *diffusivity, const float *u, int w, int h, int nc, float epsilon)
{
    // calculate block and grid size
    dim3 block(0, 0, 0);     // TODO (11.2) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // run cuda kernel
    // TODO (11.2) execute kernel for computing diffusivity

    // check for errors
    // TODO (11.2)
}


void computeDiffusionTensorCuda(float *d_difftensor11, float *d_difftensor12, float *d_difftensor22,
                                float *d_tensor11, float *d_tensor12, float *d_tensor22,
                                float alpha, float C, int w, int h, int nc)
{
    // calculate block and grid size
    dim3 block(0, 0, 0);     // TODO (10.1) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // run cuda kernel
    // TODO (10.1) execute kernel for computing diffusion tensor

    // check for errors
    // TODO (10.1)
}
