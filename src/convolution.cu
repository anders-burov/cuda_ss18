#include "hip/hip_runtime.h"
// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################
#include "convolution.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include "helper.cuh"


// TODO (6.3) define constant memory for convolution kernel
#define KERNEL_MAX 41 // max diameter for constant kernel
__constant__ float constKernel[KERNEL_MAX*KERNEL_MAX];

// TODO (6.2) define texture for image
texture<float,2,hipReadModeElementType> texRef;


__global__
void computeConvolutionTextureMemKernel(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    // TODO (6.2) compute convolution using texture memory
    int x = threadIdx.x + blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;
    float val = tex2D(texRef, x+0.5f, y+0.5f);

    int kdiameter = 2*kradius+1;

    if (x < w && y < h)
    {
        for (int z = 0; z < nc; z++)
        {
            int idx = z*h*w + y*w + x;
            imgOut[idx] = 0;
            for (int v = -kradius; v <= kradius; v++)
            {
                for (int u = -kradius; u <= kradius; u++)
                {
                   imgOut[idx] += tex2D(texRef, x+u+0.5, max(min(y+v+0.5,h-0.5),0.5) + h*z) * kernel[(v+kradius)*kdiameter+(u+kradius)];
                }
            }
        }
    }
}


__global__
void computeConvolutionSharedMemKernel(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc, int sm_x, int sm_y)
{
    // TODO (6.1) compute convolution using shared memory
    extern __shared__ float shared[];

    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int block_size = blockDim.x*blockDim.y;
    int shared_size = sm_y*sm_x;

    int kdiameter = 2*kradius+1;

    for (int z = 0; z < nc; z++)
    {
        for (int i = threadIdx.y*blockDim.x+threadIdx.x; i < shared_size; i+=block_size)
        {
            // shared coordinates
            int u = i % sm_x;
            int v = i / sm_x;

            int glob_x = u - kradius + blockIdx.x*blockDim.x;
            int glob_y = v - kradius + blockIdx.y*blockDim.y;
            shared[i] = imgIn[z*h*w + max(min(glob_y,h-1),0)*w + max(min(glob_x,w-1),0)];
        }

        // set print array on
        // set logging file log.txt
        // set logging on
        // less log.txt | tr -d ',} {$=' | awk '{ if (NR % 26 == 0) { print $0"\0" } else { print $0 } }' | tr '\n' '\t' | tr '\0' '\n' | tail -c +2 | less > out.txt

        __syncthreads();

        if (x < w && y < h)
        {
            int idx = z*h*w + y*w + x;
            imgOut[idx] = 0;

            for (int j = 0; j < kdiameter; j++)
            {
                for (int i = 0; i < kdiameter; i++)
                {
                   //kernel from the global memory
                   //imgOut[idx] += shared[(threadIdx.y+j)*sm_x+(threadIdx.x+i)] * kernel[j*kdiameter+i];

                   //kernel from the constant memory
                   imgOut[idx] += shared[(threadIdx.y+j)*sm_x+(threadIdx.x+i)] * constKernel[j*KERNEL_MAX+i];
                }
            }
        }

        __syncthreads();
	}
}


__global__
void computeConvolutionGlobalMemKernel(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    // TODO (5.4) compute convolution using global memory
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    int kdiameter = 2*kradius+1;

    if (x < w && y < h)
    {
        for (int c = 0; c < nc; c++)
        {
            int idx = c*h*w + y*w + x;
            imgOut[idx] = 0;
            for (int v = -kradius; v <= kradius; v++)
            {
                for (int u = -kradius; u <= kradius; u++)
                {
                   imgOut[idx] += imgIn[c*w*h + max(min(y+v,h-1),0)*w + max(min(x+u,w-1),0)]*kernel[(v+kradius)*kdiameter+(u+kradius)];
                }
            }
        }
    }
}


void createConvolutionKernel(float *kernel, int kradius, float sigma)
{
    // TODO (5.1) fill convolution kernel
    int kdiagonal = 2*kradius+1;
    for (int j = -kradius; j <= kradius; j++)
    {
        for (int i = -kradius; i <= kradius; i++)
        {
            kernel[(j+kradius)*kdiagonal+(i+kradius)] = expf(-(i*i+j*j)/(2*sigma*sigma))/(2*PI*sigma*sigma);
        }
    }
}


void computeConvolution(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    if (!imgOut || !imgIn)
    {
        std::cerr << "arrays not allocated!" << std::endl;
        return;
    }

    int kdiameter = 2*kradius+1;

    // TODO (5.3) compute convolution on CPU
    for (int c = 0; c < nc; c++)
    {
        for (int j = 0; j < h; j++)
        {
            for (int i = 0; i < w; i++)
            {
                int idx = c*w*h + j*w + i;
                imgOut[idx] = 0;
                for (int v = -kradius; v <= kradius; v++)
                {
                    for (int u = -kradius; u <= kradius; u++)
                    {
// Dirichlet Boundary
//                       if (j+v >= 0 && j+v < h && i+u >= 0 && i+u < w)
//                       {
//                           int conv_idx = c*w*h + (j+v)*w + (i+u);
//                           imgOut[idx] += imgIn[conv_idx]*kernel[(v+kradius)*kdiameter+(u+kradius)];
//                       }

// von Neuman Boundary
                         imgOut[idx] += imgIn[c*w*h + max(min(j+v,h-1),0)*w + max(min(i+u,w-1),0)]*kernel[(v+kradius)*kdiameter+(u+kradius)];
                    }
                }
            }
        }
    }
}


void computeConvolutionTextureMemCuda(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    if (!imgOut || !imgIn)
    {
        std::cerr << "arrays not allocated!" << std::endl;
        return;
    }

    // calculate block and grid size
    dim3 block(32, 32, 1);     // TODO (6.2) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    texRef.addressMode[0] = hipAddressModeClamp; // clamp x to border
    texRef.addressMode[1] = hipAddressModeClamp; // clamp y to border
    texRef.normalized = false; // access as (x+0.5f,y+0.5f), not as ((x+0.5f)/w,(y+0.5f)/h)

    // TODO (6.2) bind texture
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    hipBindTexture2D(NULL, &texRef, imgIn, &desc, w, h*nc, w*sizeof(imgIn[0])); CUDA_CHECK;

    // run cuda kernel
    // TODO (6.2) execute kernel for convolution using global memory
    computeConvolutionTextureMemKernel <<<grid, block>>> (imgOut, imgIn, kernel, kradius, w, h, nc);
    CUDA_CHECK;

    // TODO (6.2) unbind texture
    hipUnbindTexture(texRef);

    // check for errors
    // TODO (6.2)
    CUDA_CHECK;
}


void computeConvolutionSharedMemCuda(float *imgOut, const float *imgIn, const float *kernel, const float *kernel_cpu, int kradius, int w, int h, int nc)
{
    if (!imgOut || !imgIn)
    {
        std::cerr << "arrays not allocated!" << std::endl;
        return;
    }

    int kdiameter = 2*kradius+1;

    if (KERNEL_MAX < kdiameter)
    {
        std::cerr << "kernel diameter bigger than the allowed size!" << std::endl;
        return;
    }

    float *kernel_pitched = new float[KERNEL_MAX*KERNEL_MAX];
    for (int j = 0; j < kdiameter; j++)
    {
        for (int i = 0; i < kdiameter; i++)
        {
            kernel_pitched[j*KERNEL_MAX+i] = kernel_cpu[j*kdiameter+i];
        }
    }
    hipMemcpyToSymbol(HIP_SYMBOL(constKernel), kernel_pitched, KERNEL_MAX*KERNEL_MAX*sizeof(float)); CUDA_CHECK;

    // calculate block and grid size
    dim3 block(32, 32, 1);     // TODO (6.1) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // TODO (6.1) calculate shared memory size
    int sm_x = block.x + 2*kradius;
    int sm_y = block.y + 2*kradius;
    size_t smBytes = sm_x * sm_y * sizeof(float);

    // run cuda kernel
    // TODO (6.1) execute kernel for convolution using global memory
    computeConvolutionSharedMemKernel <<<grid, block, smBytes>>> (imgOut, imgIn, kernel, kradius, w, h, nc, sm_x, sm_y);

    // check for errors
    // TODO (6.1)
    CUDA_CHECK;

    delete[] kernel_pitched;
}


void computeConvolutionGlobalMemCuda(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    if (!imgOut || !imgIn)
    {
        std::cerr << "arrays not allocated!" << std::endl;
        return;
    }

    // calculate block and grid size
    dim3 block(32, 8, 1);     // TODO (5.4) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // run cuda kernel
    // TODO (5.4) execute kernel for convolution using global memory
    computeConvolutionGlobalMemKernel <<<grid, block>>> (imgOut, imgIn, kernel, kradius, w, h, nc);

    // check for errors
    // TODO (5.4)
    CUDA_CHECK;
}
