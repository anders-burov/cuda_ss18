#include "hip/hip_runtime.h"
// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################
#include "convolution.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include "helper.cuh"


// TODO (6.3) define constant memory for convolution kernel

// TODO (6.2) define texture for image


__global__
void computeConvolutionTextureMemKernel(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    // TODO (6.2) compute convolution using texture memory
}


__global__
void computeConvolutionSharedMemKernel(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    int id_x = threadIdx.x + blockDim.x * blockIdx.x;
    int id_y = threadIdx.y + blockDim.y * blockIdx.y;

    int kdiameter = 2*kradius+1;

    if (id_x < w && id_y < h)
    {
        for (int c = 0; c < nc; c++)
        {
            int idx = c*h*w + id_y*w + id_x;
            imgOut[idx] = 0;
            for (int v = -kradius; v <= kradius; v++)
            {
                for (int u = -kradius; u <= kradius; u++)
                {
                   imgOut[idx] += imgIn[c*w*h + max(min(id_y+v,h-1),0)*w + max(min(id_x+u,w-1),0)]*kernel[(v+kradius)*kdiameter+(u+kradius)];
                }
            }
        }
    }
}


__global__
void computeConvolutionGlobalMemKernel(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    // TODO (5.4) compute convolution using global memory
}


void createConvolutionKernel(float *kernel, int kradius, float sigma)
{
    // TODO (5.1) fill convolution kernel
    int kdiagonal = 2*kradius+1;
    for (int j = -kradius; j <= kradius; j++)
    {
        for (int i = -kradius; i <= kradius; i++)
        {
            kernel[(j+kradius)*kdiagonal+(i+kradius)] = expf(-(i*i+j*j)/(2*sigma*sigma))/(2*PI*sigma*sigma);
        }
    }
}


void computeConvolution(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    if (!imgOut || !imgIn)
    {
        std::cerr << "arrays not allocated!" << std::endl;
        return;
    }

    int kdiameter = 2*kradius+1;

    // TODO (5.3) compute convolution on CPU
    for (int c = 0; c < nc; c++)
    {
        for (int j = 0; j < h; j++)
        {
            for (int i = 0; i < w; i++)
            {
                int idx = c*w*h + j*w + i;
                imgOut[idx] = 0;
                for (int v = -kradius; v <= kradius; v++)
                {
                    for (int u = -kradius; u <= kradius; u++)
                    {
// Dirichlet Boundary
//                       if (j+v >= 0 && j+v < h && i+u >= 0 && i+u < w)
//                       {
//                           int conv_idx = c*w*h + (j+v)*w + (i+u);
//                           imgOut[idx] += imgIn[conv_idx]*kernel[(v+kradius)*kdiameter+(u+kradius)];
//                       }

// von Neuman Boundary
                         imgOut[idx] += imgIn[c*w*h + max(min(j+v,h-1),0)*w + max(min(i+u,w-1),0)]*kernel[(v+kradius)*kdiameter+(u+kradius)];
                    }
                }
            }
        }
    }
}


void computeConvolutionTextureMemCuda(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    if (!imgOut || !imgIn)
    {
        std::cerr << "arrays not allocated!" << std::endl;
        return;
    }

    // calculate block and grid size
    dim3 block(0, 0, 0);     // TODO (6.2) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // TODO (6.2) bind texture

    // run cuda kernel
    // TODO (6.2) execute kernel for convolution using global memory

    // TODO (6.2) unbind texture

    // check for errors
    // TODO (6.2)
}


void computeConvolutionSharedMemCuda(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    if (!imgOut || !imgIn)
    {
        std::cerr << "arrays not allocated!" << std::endl;
        return;
    }

    // calculate block and grid size
    dim3 block(0, 0, 0);     // TODO (6.1) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // TODO (6.1) calculate shared memory size

    // run cuda kernel
    // TODO (6.1) execute kernel for convolution using global memory

    // check for errors
    // TODO (6.1)
}


void computeConvolutionGlobalMemCuda(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    if (!imgOut || !imgIn)
    {
        std::cerr << "arrays not allocated!" << std::endl;
        return;
    }

    // calculate block and grid size
    dim3 block(32, 8, 1);     // TODO (5.4) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // run cuda kernel
    // TODO (5.4) execute kernel for convolution using global memory
    computeConvolutionGlobalMemKernel <<<grid, block>>> (imgOut, imgIn, kernel, kradius, w, h, nc);

    // check for errors
    // TODO (5.4)
    CUDA_CHECK;
}
