#include "hip/hip_runtime.h"
// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################
#include "convolution.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include "helper.cuh"


// TODO (6.3) define constant memory for convolution kernel

// TODO (6.2) define texture for image


__global__
void computeConvolutionTextureMemKernel(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    // TODO (6.2) compute convolution using texture memory
}


__global__
void computeConvolutionSharedMemKernel(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc, int sm_x, int sm_y)
{
    // TODO (6.1) compute convolution using shared memory
    extern __shared__ float shared[];

    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int block_size = blockDim.x*blockDim.y;
    int shared_size = sm_y*sm_x;

    int kdiameter = 2*kradius+1;

    for (int z = 0; z < nc; z++)
    {
        for (int i = threadIdx.y*blockDim.x+threadIdx.x; i < shared_size; i+=block_size)
        {
            // shared coordinates
            int u = i % sm_x;
            int v = i / sm_x;

            int glob_x = u - kradius + blockIdx.x*blockDim.x;
            int glob_y = v - kradius + blockIdx.y*blockDim.y;
            shared[i] = imgIn[z*h*w + max(min(glob_y,h-1),0)*w + max(min(glob_x,w-1),0)];
        }

        // set print array on
        // set logging file log.txt
        // set logging on
        // less log.txt | tr -d ',} {$=' | awk '{ if (NR % 26 == 0) { print $0"\0" } else { print $0 } }' | tr '\n' '\t' | tr '\0' '\n' | tail -c +2 | less > out.txt

        __syncthreads();

        if (x < w && y < h)
        {
            int idx = z*h*w + y*w + x;

            for (int j = 0; j < kdiameter; j++)
            {
                for (int i = 0; i < kdiameter; i++)
                {
                   imgOut[idx] += shared[(threadIdx.y+j)*sm_x+(threadIdx.x+i)] * kernel[j*kdiameter+i];
                }
            }
        }

        __syncthreads();
	}
}


__global__
void computeConvolutionGlobalMemKernel(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    // TODO (5.4) compute convolution using global memory
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    int kdiameter = 2*kradius+1;

    if (x < w && y < h)
    {
        for (int c = 0; c < nc; c++)
        {
            int idx = c*h*w + y*w + x;
            imgOut[idx] = 0;
            for (int v = -kradius; v <= kradius; v++)
            {
                for (int u = -kradius; u <= kradius; u++)
                {
                   imgOut[idx] += imgIn[c*w*h + max(min(y+v,h-1),0)*w + max(min(x+u,w-1),0)]*kernel[(v+kradius)*kdiameter+(u+kradius)];
                }
            }
        }
    }
}


void createConvolutionKernel(float *kernel, int kradius, float sigma)
{
    // TODO (5.1) fill convolution kernel
    int kdiagonal = 2*kradius+1;
    for (int j = -kradius; j <= kradius; j++)
    {
        for (int i = -kradius; i <= kradius; i++)
        {
            kernel[(j+kradius)*kdiagonal+(i+kradius)] = expf(-(i*i+j*j)/(2*sigma*sigma))/(2*PI*sigma*sigma);
        }
    }
}


void computeConvolution(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    if (!imgOut || !imgIn)
    {
        std::cerr << "arrays not allocated!" << std::endl;
        return;
    }

    int kdiameter = 2*kradius+1;

    // TODO (5.3) compute convolution on CPU
    for (int c = 0; c < nc; c++)
    {
        for (int j = 0; j < h; j++)
        {
            for (int i = 0; i < w; i++)
            {
                int idx = c*w*h + j*w + i;
                imgOut[idx] = 0;
                for (int v = -kradius; v <= kradius; v++)
                {
                    for (int u = -kradius; u <= kradius; u++)
                    {
// Dirichlet Boundary
//                       if (j+v >= 0 && j+v < h && i+u >= 0 && i+u < w)
//                       {
//                           int conv_idx = c*w*h + (j+v)*w + (i+u);
//                           imgOut[idx] += imgIn[conv_idx]*kernel[(v+kradius)*kdiameter+(u+kradius)];
//                       }

// von Neuman Boundary
                         imgOut[idx] += imgIn[c*w*h + max(min(j+v,h-1),0)*w + max(min(i+u,w-1),0)]*kernel[(v+kradius)*kdiameter+(u+kradius)];
                    }
                }
            }
        }
    }
}


void computeConvolutionTextureMemCuda(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    if (!imgOut || !imgIn)
    {
        std::cerr << "arrays not allocated!" << std::endl;
        return;
    }

    // calculate block and grid size
    dim3 block(0, 0, 0);     // TODO (6.2) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // TODO (6.2) bind texture

    // run cuda kernel
    // TODO (6.2) execute kernel for convolution using global memory

    // TODO (6.2) unbind texture

    // check for errors
    // TODO (6.2)
}


void computeConvolutionSharedMemCuda(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    if (!imgOut || !imgIn)
    {
        std::cerr << "arrays not allocated!" << std::endl;
        return;
    }

    // calculate block and grid size
    dim3 block(32, 32, 1);     // TODO (6.1) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // TODO (6.1) calculate shared memory size
    int sm_x = block.x + 2*kradius;
    int sm_y = block.y + 2*kradius;
    size_t smBytes = sm_x * sm_y * sizeof(float);

    // run cuda kernel
    // TODO (6.1) execute kernel for convolution using global memory
    computeConvolutionSharedMemKernel <<<grid, block, smBytes>>> (imgOut, imgIn, kernel, kradius, w, h, nc, sm_x, sm_y);

    // check for errors
    // TODO (6.1)
    CUDA_CHECK;
}


void computeConvolutionGlobalMemCuda(float *imgOut, const float *imgIn, const float *kernel, int kradius, int w, int h, int nc)
{
    if (!imgOut || !imgIn)
    {
        std::cerr << "arrays not allocated!" << std::endl;
        return;
    }

    // calculate block and grid size
    dim3 block(32, 8, 1);     // TODO (5.4) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // run cuda kernel
    // TODO (5.4) execute kernel for convolution using global memory
    computeConvolutionGlobalMemKernel <<<grid, block>>> (imgOut, imgIn, kernel, kradius, w, h, nc);

    // check for errors
    // TODO (5.4)
    CUDA_CHECK;
}
