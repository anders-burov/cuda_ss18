#include "hip/hip_runtime.h"
// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################
#include "divergence.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include "helper.cuh"


__global__
void computeDivergenceKernel(float *q, const float *v1, const float *v2, int w, int h, int nc)
{
    // TODO (4.2) compute divergence
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;

    if (x < w && y < h)
    {
        int idx = z*h*w + y*w + x;
        q[idx] = v1[idx] - v1[z*h*w + y*w + max(0,x-1)];
        q[idx] += v2[idx] - v2[z*h*w + max(0,y-1)*w + x];
    }
}


void computeDivergenceCuda(float *q, const float *v1, const float *v2, int w, int h, int nc)
{
    // calculate block and grid size
    dim3 block(32, 8, nc);     // TODO (4.2) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // run cuda kernel
    // TODO (4.2) execute divergence kernel
    computeDivergenceKernel <<<grid, block>>> (q, v1, v2, w, h, nc);

    // check for errors
    // TODO (4.2)
    CUDA_CHECK;
}
