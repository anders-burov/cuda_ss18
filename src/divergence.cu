#include "hip/hip_runtime.h"
// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################
#include "divergence.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include "helper.cuh"


__global__
void computeDivergenceKernel(float *q, const float *v1, const float *v2, int w, int h, int nc)
{
    // TODO (4.2) compute divergence
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;

    // x-direction
    if (x > 1 && x < w && y < h)
    {
        int idx = z*h*w + y*w + x;
        int idx0 = z*h*w + y*w + x-1;
        q[idx] = v1[idx] - v1[idx0];
    }

    // y-direction
    if (x < w && y > 1 && y < h)
    {
        int idx = z*h*w + y*w + x;
        int idx0 = z*h*w + (y-1)*w + x;
        q[idx] += v2[idx] - v2[idx0];
    }
}


void computeDivergenceCuda(float *q, const float *v1, const float *v2, int w, int h, int nc)
{
    // calculate block and grid size
    dim3 block(32, 8, nc);     // TODO (4.2) specify suitable block size
    dim3 grid = computeGrid2D(block, w, h);

    // run cuda kernel
    // TODO (4.2) execute divergence kernel
    computeDivergenceKernel <<<grid, block>>> (q, v1, v2, w, h, nc);

    // check for errors
    // TODO (4.2)
    CUDA_CHECK;
}
