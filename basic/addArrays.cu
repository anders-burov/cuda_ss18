// ########################################################################
// Practical Course: GPU Programming in Computer Vision
// Technical University of Munich, Computer Vision Group
// ########################################################################

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;



// cuda error checking
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        exit(1);
    }
}

__global__ void AddArrays(float* a, float *b, float *c, int n)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n)
    {
        c[idx] = a[idx] + b[idx];
    }
}

int main(int argc, char **argv)
{
    // alloc and init input arrays on host (CPU)
    int n = 20;
    float *a = new float[n];
    float *b = new float[n];
    float *c = new float[n];
    for(int i=0; i<n; i++)
    {
        a[i] = i;
        b[i] = (i%5)+1;
        c[i] = 0;
    }

    // CPU computation
    for(int i=0; i<n; i++) c[i] = a[i] + b[i];

    // print result
    cout << "CPU:"<<endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << " + " << b[i] << " = " << c[i] << endl;
    cout << endl;
    // init c
    for(int i=0; i<n; i++) c[i] = 0;
    
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, n * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_b, n * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_c, n * sizeof(float)); CUDA_CHECK;

    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
    hipMemcpy(d_c, c, n * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
    
    dim3 block = dim3(n, 1, 1);
    dim3 grid = dim3(1, 1, 1);
    AddArrays <<<grid, block>>> (d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    hipFree(d_a); CUDA_CHECK;
    hipFree(d_b); CUDA_CHECK;
    hipFree(d_c); CUDA_CHECK;

    // print result
    cout << "GPU:"<<endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << " + " << b[i] << " = " << c[i] << endl;
    cout << endl;

    // free CPU arrays
    delete[] a;
    delete[] b;
    delete[] c;
}



